#include "hip/hip_runtime.h"
// Kernel function to add the elements of two arrays
__global__ void add(int n, float *x, float *y)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }
  for (int i = index; i < n; i += stride)
    y[i] = x[i] + y[i];
}

float* add_cuda(float *x, float *y) {
  // Allocate Unified Memory – accessible from CPU or GPU
  hipMallocManaged(&x, N*sizeof(float));
  hipMallocManaged(&y, N*sizeof(float));

  // initialize x and y arrays on the host
//  for (int i = 0; i < N; i++) {
//    x[i] = 1.0f;
//    y[i] = 2.0f;
//  }

  // Run kernel on 1M elements on the GPU
  int blockSize = 256;
  int numBlocks = (N + blockSize - 1) / blockSize;
  add<<<numBlocks, blockSize>>>(N, x, y);

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();

  return y
}