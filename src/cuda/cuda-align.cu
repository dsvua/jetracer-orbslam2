#include "hip/hip_runtime.h"
// #include "jetracer_rscuda_utils.cuh"
#include "cuda-align.cuh"
#include "../cuda_common.h"
#include <iostream>
#include <stdio.h> //for printf

#ifdef _MSC_VER
// Add library dependencies if using VS
#pragma comment(lib, "cudart_static")
#endif

#define RS2_CUDA_THREADS_PER_BLOCK 32

namespace Jetracer
{
    template <int N>
    struct bytes
    {
        unsigned char b[N];
    };

    int calc_block_size(int pixel_count, int thread_count)
    {
        return ((pixel_count % thread_count) == 0) ? (pixel_count / thread_count) : (pixel_count / thread_count + 1);
    }

    /* Given a point in 3D space, compute the corresponding pixel coordinates in an image with no distortion or forward distortion coefficients produced by the same camera */
    __device__ static void project_point_to_pixel(float pixel[2],
                                                  const struct rs2_intrinsics *intrin,
                                                  const float point[3])
    {
        //assert(intrin->model != RS2_DISTORTION_INVERSE_BROWN_CONRADY); // Cannot project to an inverse-distorted image

        float x = point[0] / point[2], y = point[1] / point[2];

        if (intrin->model == RS2_DISTORTION_MODIFIED_BROWN_CONRADY)
        {

            float r2 = x * x + y * y;
            float f = 1 + intrin->coeffs[0] * r2 + intrin->coeffs[1] * r2 * r2 + intrin->coeffs[4] * r2 * r2 * r2;
            x *= f;
            y *= f;
            float dx = x + 2 * intrin->coeffs[2] * x * y + intrin->coeffs[3] * (r2 + 2 * x * x);
            float dy = y + 2 * intrin->coeffs[3] * x * y + intrin->coeffs[2] * (r2 + 2 * y * y);
            x = dx;
            y = dy;
        }

        if (intrin->model == RS2_DISTORTION_FTHETA)
        {
            float r = sqrtf(x * x + y * y);
            float rd = (float)(1.0f / intrin->coeffs[0] * atan(2 * r * tan(intrin->coeffs[0] / 2.0f)));
            x *= rd / r;
            y *= rd / r;
        }

        pixel[0] = x * intrin->fx + intrin->ppx;
        pixel[1] = y * intrin->fy + intrin->ppy;
    }

    /* Given pixel coordinates and depth in an image with no distortion or inverse distortion coefficients, compute the corresponding point in 3D space relative to the same camera */
    __device__ static void deproject_pixel_to_point(float point[3],
                                                    const struct rs2_intrinsics *intrin,
                                                    const float pixel[2],
                                                    float depth)
    {
        assert(intrin->model != RS2_DISTORTION_MODIFIED_BROWN_CONRADY); // Cannot deproject from a forward-distorted image
        assert(intrin->model != RS2_DISTORTION_FTHETA);                 // Cannot deproject to an ftheta image
        //assert(intrin->model != RS2_DISTORTION_BROWN_CONRADY); // Cannot deproject to an brown conrady model

        float x = (pixel[0] - intrin->ppx) / intrin->fx;
        float y = (pixel[1] - intrin->ppy) / intrin->fy;

        if (intrin->model == RS2_DISTORTION_INVERSE_BROWN_CONRADY)
        {
            float r2 = x * x + y * y;
            float f = 1 + intrin->coeffs[0] * r2 + intrin->coeffs[1] * r2 * r2 + intrin->coeffs[4] * r2 * r2 * r2;
            float ux = x * f + 2 * intrin->coeffs[2] * x * y + intrin->coeffs[3] * (r2 + 2 * x * x);
            float uy = y * f + 2 * intrin->coeffs[3] * x * y + intrin->coeffs[2] * (r2 + 2 * y * y);
            x = ux;
            y = uy;
        }
        point[0] = depth * x;
        point[1] = depth * y;
        point[2] = depth;
    }

    /* Transform 3D coordinates relative to one sensor to 3D coordinates relative to another viewpoint */
    __device__ static void transform_point_to_point(float to_point[3],
                                                    const struct rs2_extrinsics *extrin,
                                                    const float from_point[3])
    {
        to_point[0] = extrin->rotation[0] * from_point[0] + extrin->rotation[3] * from_point[1] + extrin->rotation[6] * from_point[2] + extrin->translation[0];
        to_point[1] = extrin->rotation[1] * from_point[0] + extrin->rotation[4] * from_point[1] + extrin->rotation[7] * from_point[2] + extrin->translation[1];
        to_point[2] = extrin->rotation[2] * from_point[0] + extrin->rotation[5] * from_point[1] + extrin->rotation[8] * from_point[2] + extrin->translation[2];
    }

    __device__ void kernel_transfer_pixels(int2 *mapped_pixels,
                                           const rs2_intrinsics *depth_intrin,
                                           const rs2_intrinsics *other_intrin,
                                           const rs2_extrinsics *depth_to_other,
                                           float depth_val,
                                           int depth_x,
                                           int depth_y,
                                           int block_index)
    {
        float shift = block_index ? 0.5 : -0.5;
        auto depth_size = depth_intrin->width * depth_intrin->height;
        auto mapped_index = block_index * depth_size + (depth_y * depth_intrin->width + depth_x);

        // border check is done in kernel_map_depth_to_other
        // if (mapped_index >= depth_size * 2)
        //     return;

        int2 mapped_pixel = {-1, -1};
        // Skip over depth pixels with the value of zero, we have no depth data so we will not write anything into our aligned images
        if (depth_val != 0)
        {
            //// Map the top-left corner of the depth pixel onto the other image
            float depth_pixel[2] = {depth_x + shift, depth_y + shift}, depth_point[3], other_point[3], other_pixel[2];
            deproject_pixel_to_point(depth_point,
                                     depth_intrin,
                                     depth_pixel,
                                     depth_val);
            transform_point_to_point(other_point,
                                     depth_to_other,
                                     depth_point);
            project_point_to_pixel(other_pixel,
                                   other_intrin,
                                   other_point);
            mapped_pixel.x = static_cast<int>(other_pixel[0] + 0.5f);
            mapped_pixel.y = static_cast<int>(other_pixel[1] + 0.5f);
        }

        __syncthreads();

        mapped_pixels[mapped_index] = mapped_pixel;
    }

    __global__ void kernel_map_depth_to_other(int2 *mapped_pixels,
                                              const uint16_t *depth_in,
                                              const rs2_intrinsics *depth_intrin,
                                              const rs2_intrinsics *other_intrin,
                                              const rs2_extrinsics *depth_to_other,
                                              float depth_scale)
    {

        int depth_x = blockIdx.x * blockDim.x + threadIdx.x;
        int depth_y = blockIdx.y * blockDim.y + threadIdx.y;

        int depth_pixel_index = depth_y * depth_intrin->width + depth_x;
        if (depth_x < depth_intrin->width && depth_y < depth_intrin->height)
        {
            float depth_val = depth_in[depth_pixel_index] * depth_scale;
            kernel_transfer_pixels(mapped_pixels,
                                   depth_intrin,
                                   other_intrin,
                                   depth_to_other,
                                   depth_val,
                                   depth_x,
                                   depth_y,
                                   blockIdx.z);
        }
    }

    template <int BPP>
    __global__ void kernel_other_to_depth(unsigned char *aligned,
                                          const unsigned char *other,
                                          const int2 *mapped_pixels,
                                          const rs2_intrinsics *depth_intrin,
                                          const rs2_intrinsics *other_intrin)
    {
        int depth_x = blockIdx.x * blockDim.x + threadIdx.x;
        int depth_y = blockIdx.y * blockDim.y + threadIdx.y;

        auto depth_size = depth_intrin->width * depth_intrin->height;
        int depth_pixel_index = depth_y * depth_intrin->width + depth_x;

        if (depth_pixel_index >= depth_intrin->width * depth_intrin->height)
            return;

        int2 p0 = mapped_pixels[depth_pixel_index];
        int2 p1 = mapped_pixels[depth_size + depth_pixel_index];

        if (p0.x < 0 || p0.y < 0 || p1.x >= other_intrin->width || p1.y >= other_intrin->height)
            return;

        // Transfer between the depth pixels and the pixels inside the rectangle on the other image
        auto in_other = (const bytes<BPP> *)(other);
        auto out_other = (bytes<BPP> *)(aligned);
        for (int y = p0.y; y <= p1.y; ++y)
        {
            for (int x = p0.x; x <= p1.x; ++x)
            {
                auto other_pixel_index = y * other_intrin->width + x;
                out_other[depth_pixel_index] = in_other[other_pixel_index];
            }
        }
    }

    __global__ void kernel_depth_to_other(unsigned int *aligned_out,
                                          const uint16_t *depth_in,
                                          const int2 *mapped_pixels,
                                          const rs2_intrinsics *depth_intrin,
                                          const rs2_intrinsics *other_intrin)
    {
        int depth_x = blockIdx.x * blockDim.x + threadIdx.x;
        int depth_y = blockIdx.y * blockDim.y + threadIdx.y;

        auto depth_size = depth_intrin->width * depth_intrin->height;
        int depth_pixel_index = depth_y * depth_intrin->width + depth_x;

        if (depth_x < depth_intrin->width && depth_y < depth_intrin->height)
        {
            int2 p0 = mapped_pixels[depth_pixel_index];
            int2 p1 = mapped_pixels[depth_size + depth_pixel_index];

            if (p0.x < 0 || p0.y < 0 || p1.x >= other_intrin->width || p1.y >= other_intrin->height)
                return;

            // Transfer between the depth pixels and the pixels inside the rectangle on the other image
            unsigned int new_val = depth_in[depth_pixel_index];
            // printf("p0 x:%d y:%d, p1 x:%d y:%d, depth: %d\n", p0.x, p0.y, p1.x, p1.y, new_val);
            for (int y = p0.y; y <= p1.y; ++y)
            {
                for (int x = p0.x; x <= p1.x; ++x)
                {
                    atomicMin(&aligned_out[y * other_intrin->width + x], new_val);
                }
            }
        }
    }

    __global__ void kernel_reset_to_zero(unsigned int *aligned_out,
                                         const rs2_intrinsics *other_intrin)
    {
        int x = blockIdx.x * blockDim.x + threadIdx.x;
        int y = blockIdx.y * blockDim.y + threadIdx.y;

        if (x < other_intrin->width && y < other_intrin->height)
        {
            aligned_out[y * other_intrin->width + x] = 0;
        }
    }

    void align_depth_to_other(unsigned int *d_aligned_out,
                              const uint16_t *d_depth_in,
                              int2 *d_pixel_map,
                              float depth_scale,
                              int image_width,
                              int image_height,
                              const rs2_intrinsics *d_depth_intrin,
                              const rs2_intrinsics *d_other_intrin,
                              const rs2_extrinsics *d_depth_to_other,
                              hipStream_t stream)
    {

        dim3 threads(RS2_CUDA_THREADS_PER_BLOCK, RS2_CUDA_THREADS_PER_BLOCK);
        dim3 depth_blocks(calc_block_size(image_width, threads.x), calc_block_size(image_height, threads.y));
        dim3 other_blocks(calc_block_size(image_width, threads.x), calc_block_size(image_height, threads.y));
        dim3 mapping_blocks(depth_blocks.x, depth_blocks.y, 2);

        kernel_map_depth_to_other<<<mapping_blocks, threads, 0, stream>>>(d_pixel_map,
                                                                          d_depth_in,
                                                                          d_depth_intrin,
                                                                          d_other_intrin,
                                                                          d_depth_to_other,
                                                                          depth_scale);

        kernel_reset_to_zero<<<other_blocks, threads, 0, stream>>>(d_aligned_out,
                                                                   d_other_intrin);

        kernel_depth_to_other<<<depth_blocks, threads, 0, stream>>>(d_aligned_out,
                                                                    d_depth_in,
                                                                    d_pixel_map,
                                                                    d_depth_intrin,
                                                                    d_other_intrin);
    }

}
