#include "hip/hip_runtime.h"
#include "rscuda_utils.cuh"
#include "cuda-align.cuh"
#include "../cuda_common.h"
#include <iostream>
#include <stdio.h> //for printf

#ifdef _MSC_VER
// Add library dependencies if using VS
#pragma comment(lib, "cudart_static")
#endif

#define RS2_CUDA_THREADS_PER_BLOCK 32

// using namespace rscuda;

template <int N>
struct bytes
{
    unsigned char b[N];
};

int calc_block_size(int pixel_count, int thread_count)
{
    return ((pixel_count % thread_count) == 0) ? (pixel_count / thread_count) : (pixel_count / thread_count + 1);
}

__device__ void kernel_transfer_pixels(int2 *mapped_pixels,
                                       const rs2_intrinsics *depth_intrin,
                                       const rs2_intrinsics *other_intrin,
                                       const rs2_extrinsics *depth_to_other,
                                       float depth_val,
                                       int depth_x,
                                       int depth_y,
                                       int block_index)
{
    float shift = block_index ? 0.5 : -0.5;
    auto depth_size = depth_intrin->width * depth_intrin->height;
    auto mapped_index = block_index * depth_size + (depth_y * depth_intrin->width + depth_x);

    // border check is done in kernel_map_depth_to_other
    // if (mapped_index >= depth_size * 2)
    //     return;

    int2 mapped_pixel = {-1, -1};
    // Skip over depth pixels with the value of zero, we have no depth data so we will not write anything into our aligned images
    if (depth_val != 0)
    {
        //// Map the top-left corner of the depth pixel onto the other image
        float depth_pixel[2] = {depth_x + shift, depth_y + shift}, depth_point[3], other_point[3], other_pixel[2];
        rscuda::rs2_deproject_pixel_to_point(depth_point,
                                             depth_intrin,
                                             depth_pixel,
                                             depth_val);
        rscuda::rs2_transform_point_to_point(other_point,
                                             depth_to_other,
                                             depth_point);
        rscuda::rs2_project_point_to_pixel(other_pixel,
                                           other_intrin,
                                           other_point);
        mapped_pixel.x = static_cast<int>(other_pixel[0] + 0.5f);
        mapped_pixel.y = static_cast<int>(other_pixel[1] + 0.5f);
    }

    __syncthreads();

    mapped_pixels[mapped_index] = mapped_pixel;
}

__global__ void kernel_map_depth_to_other(int2 *mapped_pixels,
                                          const uint16_t *depth_in,
                                          const rs2_intrinsics *depth_intrin,
                                          const rs2_intrinsics *other_intrin,
                                          const rs2_extrinsics *depth_to_other,
                                          float depth_scale)
{

    int depth_x = blockIdx.x * blockDim.x + threadIdx.x;
    int depth_y = blockIdx.y * blockDim.y + threadIdx.y;

    int depth_pixel_index = depth_y * depth_intrin->width + depth_x;
    if (depth_x < depth_intrin->width && depth_y < depth_intrin->height)
    {
        float depth_val = depth_in[depth_pixel_index] * depth_scale;
        kernel_transfer_pixels(mapped_pixels,
                               depth_intrin,
                               other_intrin,
                               depth_to_other,
                               depth_val,
                               depth_x,
                               depth_y,
                               blockIdx.z);
    }
}

template <int BPP>
__global__ void kernel_other_to_depth(unsigned char *aligned,
                                      const unsigned char *other,
                                      const int2 *mapped_pixels,
                                      const rs2_intrinsics *depth_intrin,
                                      const rs2_intrinsics *other_intrin)
{
    int depth_x = blockIdx.x * blockDim.x + threadIdx.x;
    int depth_y = blockIdx.y * blockDim.y + threadIdx.y;

    auto depth_size = depth_intrin->width * depth_intrin->height;
    int depth_pixel_index = depth_y * depth_intrin->width + depth_x;

    if (depth_pixel_index >= depth_intrin->width * depth_intrin->height)
        return;

    int2 p0 = mapped_pixels[depth_pixel_index];
    int2 p1 = mapped_pixels[depth_size + depth_pixel_index];

    if (p0.x < 0 || p0.y < 0 || p1.x >= other_intrin->width || p1.y >= other_intrin->height)
        return;

    // Transfer between the depth pixels and the pixels inside the rectangle on the other image
    auto in_other = (const bytes<BPP> *)(other);
    auto out_other = (bytes<BPP> *)(aligned);
    for (int y = p0.y; y <= p1.y; ++y)
    {
        for (int x = p0.x; x <= p1.x; ++x)
        {
            auto other_pixel_index = y * other_intrin->width + x;
            out_other[depth_pixel_index] = in_other[other_pixel_index];
        }
    }
}

__global__ void kernel_depth_to_other(unsigned int *aligned_out,
                                      const uint16_t *depth_in,
                                      const int2 *mapped_pixels,
                                      const rs2_intrinsics *depth_intrin,
                                      const rs2_intrinsics *other_intrin)
{
    int depth_x = blockIdx.x * blockDim.x + threadIdx.x;
    int depth_y = blockIdx.y * blockDim.y + threadIdx.y;

    auto depth_size = depth_intrin->width * depth_intrin->height;
    int depth_pixel_index = depth_y * depth_intrin->width + depth_x;

    if (depth_x < depth_intrin->width && depth_y < depth_intrin->height)
    {
        int2 p0 = mapped_pixels[depth_pixel_index];
        int2 p1 = mapped_pixels[depth_size + depth_pixel_index];

        if (p0.x < 0 || p0.y < 0 || p1.x >= other_intrin->width || p1.y >= other_intrin->height)
            return;

        // Transfer between the depth pixels and the pixels inside the rectangle on the other image
        unsigned int new_val = depth_in[depth_pixel_index];
        // printf("p0 x:%d y:%d, p1 x:%d y:%d, depth: %d\n", p0.x, p0.y, p1.x, p1.y, new_val);
        for (int y = p0.y; y <= p1.y; ++y)
        {
            for (int x = p0.x; x <= p1.x; ++x)
            {
                atomicMin(&aligned_out[y * other_intrin->width + x], new_val);
            }
        }
    }
}

__global__ void kernel_reset_to_zero(unsigned int *aligned_out,
                                     const rs2_intrinsics *other_intrin)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < other_intrin->width && y < other_intrin->height)
    {
        aligned_out[y * other_intrin->width + x] = 0;
    }
}

void align_depth_to_other(unsigned int *d_aligned_out,
                          const uint16_t *d_depth_in,
                          int2 *d_pixel_map,
                          float depth_scale,
                          int image_width,
                          int image_height,
                          const rs2_intrinsics *d_depth_intrin,
                          const rs2_intrinsics *d_other_intrin,
                          const rs2_extrinsics *d_depth_to_other,
                          hipStream_t stream)
{

    dim3 threads(RS2_CUDA_THREADS_PER_BLOCK, RS2_CUDA_THREADS_PER_BLOCK);
    dim3 depth_blocks(calc_block_size(image_width, threads.x), calc_block_size(image_height, threads.y));
    dim3 other_blocks(calc_block_size(image_width, threads.x), calc_block_size(image_height, threads.y));
    dim3 mapping_blocks(depth_blocks.x, depth_blocks.y, 2);

    kernel_map_depth_to_other<<<mapping_blocks, threads, 0, stream>>>(d_pixel_map,
                                                                      d_depth_in,
                                                                      d_depth_intrin,
                                                                      d_other_intrin,
                                                                      d_depth_to_other,
                                                                      depth_scale);

    kernel_reset_to_zero<<<other_blocks, threads, 0, stream>>>(d_aligned_out,
                                                               d_other_intrin);

    kernel_depth_to_other<<<depth_blocks, threads, 0, stream>>>(d_aligned_out,
                                                                d_depth_in,
                                                                d_pixel_map,
                                                                d_depth_intrin,
                                                                d_other_intrin);
}
